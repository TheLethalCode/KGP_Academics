
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

bool cudaCheck(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "Code Failed due to " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    return true;   
}

void printProp(hipDeviceProp_t devP) {
    std::cout << "Name: " << devP.name << std::endl;
    std::cout << "\tTotal Global Memory: " << devP.totalGlobalMem << std::endl;
    std::cout << "\tShared Memory per Block: " << devP.sharedMemPerBlock << std::endl;
    std::cout << "\tWarp Size: " << devP.warpSize << std::endl;
    std::cout << "\tMax Threads per Block: " << devP.maxThreadsPerBlock << std::endl;
    std::cout << "\tNumber of multiprocessors: " << devP.multiProcessorCount << std::endl;
    for (int i = 0; i < 3; i++) {
        std::cout << "\tMax of dimension " << i << " of block: " << devP.maxThreadsDim[i] << std::endl;
    }
    for (int i = 0; i < 3; i++) {
         std::cout << "\tMax of dimension " << i << " of grid: " << devP.maxGridSize[i] << std::endl;
    }
}

void DeviceProp() {
    int devCount ;
    hipGetDeviceCount(&devCount) ;
    for (int i = 0; i < devCount ; ++i) {
        hipDeviceProp_t devP;
        hipGetDeviceProperties(&devP, i);
        printProp(devP);
    }
}

int vecToArr(std::vector< int > &v, int **A) {
    *A = new int[v.size()];
    for (int i = 0; i < v.size(); i++) {
        (*A)[i] = v[i];
    }
    return v.size();
}
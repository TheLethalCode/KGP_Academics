#include "../include/core.h"
#include "../include/graph.h"
#include "../include/BFSutils.h"
#include "../include/algoCPU.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <utility>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 1024

int main(int argc, char* argv[]) {

    int s;
    Graph G(argc, argv);
    // G.printGraph();
    std::cout << "Source Vertex: ";
    std::cin >> s;
    // ========================= CUDA ============================= //
      
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Declare and Initialise Host Array
    int N, level, queueSize, Es;
    int *V, *E, *D, *P;
    N = vecToArr(G.posV, &V);
    Es = vecToArr(G.packE, &E);
    D = new int[N];
    P = new int[N];
    std::fill_n(D, N, INF);
    std::fill_n(P, N, -1);
    D[s] = level = 0; // Update source values
    queueSize = 1;

    int *nextQueueSize;
    hipHostMalloc((void **) &nextQueueSize, sizeof(int));
    *nextQueueSize = 0;
    // Declare and Initialise Device Array
    int *devV, *devE, *devD, *devP;
    int *devCurrentQueue, *devNextQueue;
    allocCopy<int>(&devV, V, N, "V_a");
    allocCopy<int>(&devE, E, Es, "E_a");
    allocCopy<int>(&devD, D, N, "D_a");
    allocCopy<int>(&devP, P, N, "P_a");
    alloc<int>(&devCurrentQueue, N, "devCurrentQueue");
    alloc<int>(&devNextQueue, N, "devNextQueue");

    int firstElemQueue = s;
    hipMemcpy(devCurrentQueue, &firstElemQueue, sizeof(int), hipMemcpyHostToDevice);
    
    // Run Cuda Parallel
    int blocks = (N + NUM_THREADS - 1) / NUM_THREADS;
    hipEventRecord(start);
    while (queueSize) {
        queueBfs<<< blocks, NUM_THREADS >>>(level, devV, devE, devD, devP, queueSize, nextQueueSize, devCurrentQueue, devNextQueue);
        hipDeviceSynchronize();
        level += 1;
        queueSize = *nextQueueSize;
        *nextQueueSize = 0;
        std::swap(devCurrentQueue, devNextQueue);
    }
    hipEventRecord(stop);
    cudaCheck(hipPeekAtLastError());
    if (cudaCheck(hipMemcpy(D, devD, N * sizeof(int), hipMemcpyDeviceToHost))) {
        std::cout << "Obtained distance in host at D_a" << std::endl;
    }

    // Calculate Time Taken
    hipEventSynchronize(stop);
    float timeGPU = 0;
    hipEventElapsedTime(&timeGPU, start, stop);

    // Free memory
    clear<int>(devV, "devV");
    clear<int>(devE, "devE");
    clear<int>(devD, "devD");
    clear<int>(devP, "devP");
    clear<int>(devCurrentQueue, "devCurrentQueue");
    clear<int>(devNextQueue, "devNextQueue");
    
    // ========================= CPU ============================= //
    int *dis = new int[N];
    int *parent = new int[N];
    bool *visited = new bool[N];
    std::fill_n(dis, N, INF);
    std::fill_n(parent, N, -1);
    std::fill_n(visited, N, false);
    auto beg = std::chrono::high_resolution_clock::now();
    bfsCPU(s, G, dis, parent, visited);
    auto end = std::chrono::high_resolution_clock::now();
    float timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();

    std::cout << "CUDA Elapsed Time (in ms): " << timeGPU << std::endl;
    std::cout << "CPU  Elapsed Time (in ms): " << timeCPU / 1000 << std::endl;

    // ======================= Verification ==========================//
    for (int i = 0; i < N; i++) {
        if (dis[i] != D[i]) {
            std::cout << "Not a Match at " << i << std::endl;
            std::cout << "GPU dist: " << D[i] << std::endl;
            std::cout << "CPU dist: " << dis[i] << std::endl;
            exit(EXIT_FAILURE);
        }
    }
}

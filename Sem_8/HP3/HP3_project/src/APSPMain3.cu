#include "../include/core.h"
#include "../include/graph.h"
#include "../include/APSPutils.h"
#include "../include/algoCPU.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <utility>

int main(int argc, char* argv[]) {
    hipEvent_t start, stop;

    Graph G(argc, argv);
    std::cout << "Graph Generated " << std::endl;
    // ========================= CUDA ============================= //
      
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Declare and Initialise Host Array
    int **dis, *adj, size = G.n + 1;
    adj = new int[size*size];
    dis = new int*[size];
    for (int i = 0; i <= G.n; i++) {
        dis[i] = new int[size];
        for (int j = 0; j <= G.n; j++) {
            adj[i*size + j] = dis[i][j] = INF;
        }
        adj[i*size + i] = dis[i][i] = 0;
        for (auto v : G.E[i]) {
            adj[i*size + v.first] = dis[i][v.first] = v.second;
        }
    }

    // Declare and Initialise Device Array
    int *graphDevice;
    size_t height = size;
    size_t width = height*sizeof(int);
    size_t pitch;

    cudaCheck(hipMallocPitch(&graphDevice, &pitch, width, height));
    cudaCheck(hipMemcpy2D(graphDevice, pitch, adj, width, width, height, hipMemcpyHostToDevice));

    dim3 gridPhase1(1 ,1, 1);
    dim3 gridPhase2((size - 1) / BLOCK_SIZE + 1, 2 , 1);
    dim3 gridPhase3((size - 1) / BLOCK_SIZE + 1, (size - 1) / BLOCK_SIZE + 1 , 1);
    dim3 dimBlockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
    int numBlock = (size - 1) / BLOCK_SIZE + 1;

    hipEventRecord(start);
    for(int blockID = 0; blockID < numBlock; ++blockID) {
        // Start dependent phase
        _blocked_fw_independent_ph<<<gridPhase1, dimBlockSize>>>(blockID, pitch / sizeof(int), size, graphDevice);

        // Start partially dependent phase
        _blocked_fw_partial_dependent_ph<<<gridPhase2, dimBlockSize>>>(blockID, pitch / sizeof(int), size, graphDevice);

        // Start independent phase
        _blocked_fw_double_dependent_ph<<<gridPhase3, dimBlockSize>>>(blockID, pitch / sizeof(int), size, graphDevice);
    }
    hipEventRecord(stop);
    cudaCheck(hipPeekAtLastError());

    if(cudaCheck(hipMemcpy2D(adj, width, graphDevice, pitch, width, height, hipMemcpyDeviceToHost))){
        std::cout << "Obtained distance in host at adj" << std::endl;
    }
    cudaCheck(hipFree(graphDevice));

    // Time Calculation
    hipEventSynchronize(stop);
    float timeGPU = 0;
    hipEventElapsedTime(&timeGPU, start, stop);
    std::cout << "CUDA Elapsed Time (in ms): " << timeGPU << std::endl;

    // ========================= CPU ============================= //
    auto beg = std::chrono::high_resolution_clock::now();
    floydWarshallCPU(G.n, dis);
    auto end = std::chrono::high_resolution_clock::now();
    float timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();
    std::cout << "CPU Elapsed Time (in ms): " << timeCPU / 1000 << std::endl;

    // ======================= Verification ==========================//
    for (int i = 0; i <= G.n; i++) {
        for (int j = 0; j <= G.n; j++) {
            if (dis[i][j] != adj[i*size + j]) {
                std::cout << "Not a Match at " << i << " " << j << std::endl;
                std::cout << "GPU dist: " << adj[i*size + j] << std::endl;
                std::cout << "CPU dist: " << dis[i][j] << std::endl;
                exit(EXIT_FAILURE);
            }   
        }
    }
}
#include "../include/core.h"
#include "../include/graph.h"
#include "../include/SSSPutils.h"
#include "../include/algoCPU.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <utility>

#define NUM_THREADS 256

int main(int argc, char* argv[]) {

    Graph G(argc, argv);
    std::cout << "Graph Created " << std::endl;
    // ========================= CUDA ============================= //
      
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Declare and Initialise Host Array
    int *V, *E, *W, **C, **U, Vs, Es;
    bool **M, flag;
    Vs = vecToArr(G.posV, &V);
    Es = vecToArr(G.packE, &E);
    vecToArr(G.packW, &W);
    C = new int*[Vs];
    U = new int*[Vs];
    M = new bool*[Vs];
    for (int i = 0; i < Vs; i++) {
        C[i] = new int[Vs];
        U[i] = new int[Vs];
        M[i] = new bool[Vs];
        std::fill_n(C[i], Vs, INF);
        std::fill_n(U[i], Vs, INF);
        std::fill_n(M[i], Vs, false);
        C[i][i] = U[i][i] = 0;
        M[i][i] = true;
    }
    
    // Declare and Initialise Device Array
    int *devV, *devE, *devW, *devC, *devU;
    bool *devM, *devFlag;
    allocCopy<int>(&devV, V, Vs, "V_a");
    allocCopy<int>(&devE, E, Es, "E_a");
    allocCopy<int>(&devW, W, Es, "W_a");
    alloc<int>(&devC, Vs, "C_a");
    alloc<int>(&devU, Vs, "U_a");
    alloc<bool>(&devM, Vs, "M_a");
    alloc<bool>(&devFlag, 1, "flag");
    
    // Run Cuda Parallel
    dim3 blocks((Vs + NUM_THREADS - 1) / NUM_THREADS, 1, 1);
    dim3 threads(NUM_THREADS, 1, 1);
    hipEventRecord(start);
    for (int i = 0; i < Vs; i++) {
        flag = true;
        cudaCheck(hipMemcpy(devC, C[i], Vs*sizeof(int), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(devU, U[i], Vs*sizeof(int), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(devM, M[i], Vs*sizeof(bool), hipMemcpyHostToDevice));
        while (flag) {
            flag = false;
            hipMemcpy(devFlag, &flag, sizeof(bool), hipMemcpyHostToDevice);
            SSSP_kernel1<<< blocks, threads >>>(devV, devE, devW, devM, devC, devU, Vs);
            SSSP_kernel2<<< blocks, threads >>>(devM, devC, devU, devFlag, Vs);
            hipMemcpy(&flag, devFlag, sizeof(bool), hipMemcpyDeviceToHost);
        }
        cudaCheck(hipMemcpy(C[i], devC, Vs*sizeof(int), hipMemcpyDeviceToHost));
    }
    hipEventRecord(stop);
    cudaCheck(hipPeekAtLastError());
    std::cout << "Obtained distance in host at C_a" << std::endl;

    // Free Memory
    clear<int>(devV, "devV");
    clear<int>(devE, "devE");
    clear<int>(devW, "devW");
    clear<int>(devC, "devC");
    clear<int>(devU, "devU");
    clear<bool>(devM, "devM");
    clear<bool>(devFlag, "devFlag");

    // Calculate Time Taken
    hipEventSynchronize(stop);
    float timeGPU = 0;
    hipEventElapsedTime(&timeGPU, start, stop);
    std::cout << "CUDA Elapsed Time (in ms): " << timeGPU << std::endl;

    // ========================= CPU ============================= //
    int **dis = new int*[Vs];
    auto beg = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < Vs; i++) {
        dis[i] = new int[Vs];
        std::fill_n(dis[i], Vs, INF);
        djikstraCPU(G, i, dis[i]);
    }
    auto end = std::chrono::high_resolution_clock::now();
    float timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();
    std::cout << "CPU Elapsed Time (in ms): " << timeCPU / 1000 << std::endl;

    // ======================= Verification ==========================//
    for (int i = 0; i < Vs; i++) {
        for (int j = 0; j < Vs; j++) {
            if (dis[i][j] != C[i][j]) {
                std::cout << "Not a Match at " << i << " " << j << std::endl;
                std::cout << "GPU dist: " << C[i][j] << std::endl;
                std::cout << "CPU dist: " << dis[i][j] << std::endl;
                exit(EXIT_FAILURE);
            }
        }
    }
}
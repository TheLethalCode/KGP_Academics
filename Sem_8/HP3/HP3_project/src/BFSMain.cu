#include "../include/core.h"
#include "../include/graph.h"
#include "../include/BFSutils.h"
#include "../include/algoCPU.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <utility>

#define NUM_THREADS 1024

int main(int argc, char* argv[]) {

    int s;
    Graph G(argc, argv);
    std::cout << "Source Vertex: ";
    std::cin >> s;
    // ========================= CUDA ============================= //
      
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Declare and Initialise Host Array
    int N, level, flag, Es;
    int *V, *E, *D, *P;
    N = vecToArr(G.posV, &V);
    Es = vecToArr(G.packE, &E);
    D = new int[N];
    P = new int[N];
    std::fill_n(D, N, INF);
    std::fill_n(P, N, -1);
    D[s] = level = 0; // Update source values
    flag = 1;

    // Declare and Initialise Device Array
    int *devFlag;
    int *devV, *devE, *devD, *devP;
    allocCopy<int>(&devV, V, N, "V_a");
    allocCopy<int>(&devE, E, Es, "E_a");
    allocCopy<int>(&devD, D, N, "D_a");
    allocCopy<int>(&devP, P, N, "P_a");
    allocCopy<int>(&devFlag, &flag, 1, "flag");
    
    // Run Cuda Parallel
    int blocks = (N + NUM_THREADS - 1) / NUM_THREADS;
    hipEventRecord(start);
    while (flag) {
        flag = 0;
        hipMemcpy(devFlag, &flag, sizeof(int), hipMemcpyHostToDevice);
        BFS_kernel<<< blocks, NUM_THREADS >>>(N, level, devV, devE, devD, devP, devFlag);
        level += 1;
        hipMemcpy(&flag, devFlag, sizeof(int), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);
    cudaCheck(hipPeekAtLastError());
    if (cudaCheck(hipMemcpy(D, devD, N * sizeof(int), hipMemcpyDeviceToHost))) {
        std::cout << "Obtained distance in host at D_a" << std::endl;
    }

    // Calculate Time Taken
    hipEventSynchronize(stop);
    float timeGPU = 0;
    hipEventElapsedTime(&timeGPU, start, stop);

    // Free memory
    clear<int>(devV, "devV");
    clear<int>(devE, "devE");
    clear<int>(devD, "devD");
    clear<int>(devP, "devP");
    clear<int>(devFlag, "devFlag");

    // ========================= CPU ============================= //
    int *dis = new int[N];
    int *parent = new int[N];
    bool *visited = new bool[N];
    std::fill_n(dis, N, INF);
    std::fill_n(parent, N, -1);
    std::fill_n(visited, N, false);
    auto beg = std::chrono::high_resolution_clock::now();
    bfsCPU(s, G, dis, parent, visited);
    auto end = std::chrono::high_resolution_clock::now();
    float timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();

    std::cout << "CUDA Elapsed Time (in ms): " << timeGPU << std::endl;
    std::cout << "CPU  Elapsed Time (in ms): " << timeCPU / 1000 << std::endl;

    // ======================= Verification ==========================//
    for (int i = 0; i < N; i++) {
        if (dis[i] != D[i]) {
            std::cout << "Not a Match at " << i << std::endl;
            std::cout << "GPU dist: " << D[i] << std::endl;
            std::cout << "CPU dist: " << dis[i] << std::endl;
            exit(EXIT_FAILURE);
        }
    }
}
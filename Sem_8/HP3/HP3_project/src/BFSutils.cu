#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "../include/graph.h"

// ========================= Parallel BFS ============================= //

__global__ void BFS_kernel(int N, int level, int *devV, int *devE, int *devD, int *devP, int *devFlag) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int valueChange = 0;
    if (thid < N && devD[thid] == level) {
        int u = thid;
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int v = devE[devV[u]+i];
            if (level + 1 < devD[v]) {
                devD[v] = level + 1;
                devP[v] = i;
                valueChange = 1;
            }
        }
    }
    if (valueChange) {
        *devFlag = 1;
    }
}

// ========================= Queue BFS ============================ //

__global__ void queueBfs(int level, int *devV, int *devE, int *devD, int *devP,
              int queueSize, int *nextQueueSize, int *devCurrentQueue, int *devNextQueue) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int u = devCurrentQueue[thid];
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int v = devE[devV[u]+i];
            if (devD[v] == INF && atomicMin(&devD[v], level + 1) == INF) {
                devP[v] = u;
                int position = atomicAdd(nextQueueSize, 1);
                devNextQueue[position] = v;
            }
        }
    }
}


// ========================= Scan BFS ============================= //

__global__ void nextLayer(int level, int *devV, int *devE, int *devP, int *devD, int queueSize, int *devCurrentQueue) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if (thid < queueSize) {
        int u = devCurrentQueue[thid];
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int pos = devV[u]+i;
            int v = devE[pos];
            if (level + 1 < devD[v]) {
                devD[v] = level + 1;
                devP[v] = pos;
            }
        }
    }

}

__global__ void countDegrees(int *devV, int *devE, int *devP, int queueSize, int *devCurrentQueue, int *devDegrees) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if (thid < queueSize) {
        int u = devCurrentQueue[thid];
        int degree = 0;
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int pos = devV[u]+i;
            int v = devE[pos];
            if (devP[v] == (pos) && v != u) {
                ++degree;
            }
        }
        devDegrees[thid] = degree;
    }
}

__global__ void scanDegrees(int N, int *devDegrees, int *incrDegrees) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if (thid < N) {
        //write initial values to shared memory
        __shared__ int prefixSum[1024];
        int modulo = threadIdx.x;
        prefixSum[modulo] = devDegrees[thid];
        __syncthreads();

        //calculate scan on this block
        //go up
        for (int nodeSize = 2; nodeSize <= 1024; nodeSize <<= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < N) {
                    int nextPosition = modulo + (nodeSize >> 1);
                    prefixSum[modulo] += prefixSum[nextPosition];
                }
            }
            __syncthreads();
        }

        //write information for increment prefix sums
        if (modulo == 0) {
            int block = thid >> 10;
            incrDegrees[block + 1] = prefixSum[modulo];
        }

        //go down
        for (int nodeSize = 1024; nodeSize > 1; nodeSize >>= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < N) {
                    int next_position = modulo + (nodeSize >> 1);
                    int tmp = prefixSum[modulo];
                    prefixSum[modulo] -= prefixSum[next_position];
                    prefixSum[next_position] = tmp;

                }
            }
            __syncthreads();
        }
        devDegrees[thid] = prefixSum[modulo];
    }
}

__global__ void assignVerticesNextQueue(int *devV, int *devE, int *devP, int queueSize,
                             int *devCurrentQueue, int *devNextQueue, int *devDegrees, int *incrDegrees,
                             int nextQueueSize) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("assignVerticesNextQ thid %d\n", thid);
    if (thid < queueSize) {
        __shared__ int sharedIncrement;
        if (!threadIdx.x) {
            sharedIncrement = incrDegrees[thid >> 10];
        }
        __syncthreads();

        int sum = 0;
        if (threadIdx.x) {
            sum = devDegrees[thid - 1];
        }

        int u = devCurrentQueue[thid];
        int counter = 0;
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int v = devE[devV[u]+i];
            if (devP[v] == devV[u]+i && v != u) {
                int nextQueuePlace = sharedIncrement + sum + counter;
                devNextQueue[nextQueuePlace] = v;
                counter++;
            }
        }
    }
}
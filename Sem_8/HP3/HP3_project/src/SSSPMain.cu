#include "../include/core.h"
#include "../include/graph.h"
#include "../include/SSSPutils.h"
#include "../include/algoCPU.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <utility>

#define NUM_THREADS 256

int main(int argc, char* argv[]) {

    int s;
    Graph G(argc, argv);
    std::cout << "Source Vertex: ";
    std::cin >> s;

    // ========================= CUDA ============================= //
      
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Declare and Initialise Host Array
    int *V, *E, *W, *C, *U, Vs, Es;
    bool *M, flag;
    Vs = vecToArr(G.posV, &V);
    Es = vecToArr(G.packE, &E);
    vecToArr(G.packW, &W);
    C = new int[Vs];
    U = new int[Vs];
    M = new bool[Vs];
    std::fill_n(C, Vs, INF);
    std::fill_n(U, Vs, INF);
    std::fill_n(M, Vs, false);
    C[s] = U[s] = 0; // Update source values
    M[s] = flag = true;

    // Declare and Initialise Device Array
    int *devV, *devE, *devW, *devC, *devU;
    bool *devM, *devFlag; 
    allocCopy<int>(&devV, V, Vs, "V_a");
    allocCopy<int>(&devE, E, Es, "E_a");
    allocCopy<int>(&devW, W, Es, "W_a");
    allocCopy<int>(&devC, C, Vs, "C_a");
    allocCopy<int>(&devU, U, Vs, "U_a");
    allocCopy<bool>(&devM, M, Vs, "M_a");
    allocCopy<bool>(&devFlag, &flag, 1, "flag");
    
    // Run Cuda Parallel
    int blocks = (Vs + NUM_THREADS - 1) / NUM_THREADS;
    hipEventRecord(start);
    while (flag) {
        flag = false;
        hipMemcpy(devFlag, &flag, sizeof(bool), hipMemcpyHostToDevice);
        SSSP_kernel1<<< blocks, NUM_THREADS >>>(devV, devE, devW, devM, devC, devU, Vs);
        SSSP_kernel2<<< blocks, NUM_THREADS >>>(devM, devC, devU, devFlag, Vs);
        hipMemcpy(&flag, devFlag, sizeof(bool), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);
    cudaCheck(hipPeekAtLastError());
    if (cudaCheck(hipMemcpy(C, devC, Vs * sizeof(int), hipMemcpyDeviceToHost))) {
        std::cout << "Obtained distance in host at C_a" << std::endl;
    }

    // Free memory
    clear<int>(devV, "devV");
    clear<int>(devE, "devE");
    clear<int>(devW, "devW");
    clear<int>(devC, "devC");
    clear<int>(devU, "devU");
    clear<bool>(devM, "devM");
    clear<bool>(devFlag, "devFlag");

    // Calculate Time Taken
    hipEventSynchronize(stop);
    float timeGPU = 0;
    hipEventElapsedTime(&timeGPU, start, stop);
    std::cout << "CUDA Elapsed Time (in ms): " << timeGPU << std::endl;

    // ========================= CPU ============================= //
    int *dis = new int[Vs];
    std::fill_n(dis, Vs, INF);
    auto beg = std::chrono::high_resolution_clock::now();
    djikstraCPU(G, s, dis);
    auto end = std::chrono::high_resolution_clock::now();
    float timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();
    std::cout << "CPU Elapsed Time (in ms): " << timeCPU / 1000 << std::endl;

    // ======================= Verification ==========================//
    for (int i = 0; i < Vs; i++) {
        if (dis[i] != C[i]) {
            std::cout << "Not a Match at " << i << std::endl;
            std::cout << "GPU dist: " << C[i] << std::endl;
            std::cout << "CPU dist: " << dis[i] << std::endl;
            exit(EXIT_FAILURE);
        }
    }
}
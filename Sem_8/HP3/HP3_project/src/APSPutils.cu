#include "hip/hip_runtime.h"
#include "../include/APSPutils.h"
#include "../include/graph.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__global__ void APSP_kernel1(int *dis, int k, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if (idx < n && idy < n) {
        dis[idx*n + idy] = min(dis[idx*n + idy], dis[idx*n + k] + dis[k*n + idy]);
    }
}

// ===================== BLOCKED FW ===================================== //


__global__
void _blocked_fw_independent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph) {
    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE];

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int v1 = BLOCK_SIZE * blockId + idy;
    const int v2 = BLOCK_SIZE * blockId + idx;

    int newPath;

    const int cellId = v1 * pitch + v2;
    if (v1 < nvertex && v2 < nvertex) {
        cacheGraph[idy][idx] = graph[cellId];
    } 
    else {
        cacheGraph[idy][idx] = INF;
    }

    // Synchronize to make sure the all value are loaded in block
    __syncthreads();

    #pragma unroll
    for (int u = 0; u < BLOCK_SIZE; ++u) {
        newPath = cacheGraph[idy][u] + cacheGraph[u][idx];

        // Synchronize before calculate new value
        __syncthreads();
        if (newPath < cacheGraph[idy][idx]) {
            cacheGraph[idy][idx] = newPath;
        }

        // Synchronize to make sure that all value are current
        __syncthreads();
    }

    if (v1 < nvertex && v2 < nvertex) {
        graph[cellId] = cacheGraph[idy][idx];
    }
}


__global__
void _blocked_fw_partial_dependent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph)
{
    if (blockIdx.x == blockId) return;

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    int v1 = BLOCK_SIZE * blockId + idy;
    int v2 = BLOCK_SIZE * blockId + idx;

    __shared__ int cacheGraphBase[BLOCK_SIZE][BLOCK_SIZE];

    // Load base block for graph and predecessors
    int cellId = v1 * pitch + v2;

    if (v1 < nvertex && v2 < nvertex) {
        cacheGraphBase[idy][idx] = graph[cellId];
    } 
    else {
        cacheGraphBase[idy][idx] = INF;
    }

    // Load i-aligned singly dependent blocks
    if (blockIdx.y == 0) {
        v2 = BLOCK_SIZE * blockIdx.x + idx;
    } 
    else {
    // Load j-aligned singly dependent blocks
        v1 = BLOCK_SIZE * blockIdx.x + idy;
    }

    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE];

    // Load current block for graph and predecessors
    int currentPath;

    cellId = v1 * pitch + v2;
    if (v1 < nvertex && v2 < nvertex) {
        currentPath = graph[cellId];
    } 
    else {
        currentPath = INF;
    }
    cacheGraph[idy][idx] = currentPath;

    // Synchronize to make sure that all value are saved in cache
    __syncthreads();

    int newPath;
    // Compute i-aligned singly dependent blocks
    if (blockIdx.y == 0) {
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            newPath = cacheGraphBase[idy][u] + cacheGraph[u][idx];

            if (newPath < currentPath) {
                currentPath = newPath;
            }
            // Synchronize to make sure that all threads compare new value with old
            __syncthreads();

           // Update new values
            cacheGraph[idy][idx] = currentPath;

           // Synchronize to make sure that all threads update cache
            __syncthreads();
        }
    } 
    else {
    // Compute j-aligned singly dependent blocks
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            newPath = cacheGraph[idy][u] + cacheGraphBase[u][idx];

            if (newPath < currentPath) {
                currentPath = newPath;
            }

            // Synchronize to make sure that all threads compare new value with old
            __syncthreads();

           // Update new values
            cacheGraph[idy][idx] = currentPath;

           // Synchronize to make sure that all threads update cache
            __syncthreads();
        }
    }

    if (v1 < nvertex && v2 < nvertex) {
        graph[cellId] = currentPath;
    }
}


__global__
void _blocked_fw_double_dependent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph) {
    
    if (blockIdx.x == blockId || blockIdx.y == blockId) return;

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int v1 = blockDim.y * blockIdx.y + idy;
    const int v2 = blockDim.x * blockIdx.x + idx;

    __shared__ int cacheGraphBaseRow[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int cacheGraphBaseCol[BLOCK_SIZE][BLOCK_SIZE];

    int v1Row = BLOCK_SIZE * blockId + idy;
    int v2Col = BLOCK_SIZE * blockId + idx;

    // Load data for block
    int cellId;
    if (v1Row < nvertex && v2 < nvertex) {
        cellId = v1Row * pitch + v2;
        cacheGraphBaseRow[idy][idx] = graph[cellId];
    }
    else {
        cacheGraphBaseRow[idy][idx] = INF;
    }

    if (v1  < nvertex && v2Col < nvertex) {
        cellId = v1 * pitch + v2Col;
        cacheGraphBaseCol[idy][idx] = graph[cellId];
    }
    else {
        cacheGraphBaseCol[idy][idx] = INF;
    }

    // Synchronize to make sure the all value are loaded in virtual block
   __syncthreads();

   int currentPath;
   int newPath;

   // Compute data for block
   if (v1  < nvertex && v2 < nvertex) {
       cellId = v1 * pitch + v2;
       currentPath = graph[cellId];

        #pragma unroll
       for (int u = 0; u < BLOCK_SIZE; ++u) {
           newPath = cacheGraphBaseCol[idy][u] + cacheGraphBaseRow[u][idx];
           if (currentPath > newPath) {
               currentPath = newPath;
           }
       }
       graph[cellId] = currentPath;
   }
}
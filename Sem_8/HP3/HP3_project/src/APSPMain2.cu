#include "../include/core.h"
#include "../include/graph.h"
#include "../include/APSPutils.h"
#include "../include/algoCPU.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <utility>

#define THREADX 16

int main(int argc, char* argv[]) {

    Graph G(argc, argv);
    std::cout << "Graph Generated " << std::endl;
    // ========================= CUDA ============================= //
      
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Declare and Initialise Host Array
    int **dis, *adj, size = G.n + 1;
    adj = new int[size*size];
    dis = new int*[size];
    for (int i = 0; i <= G.n; i++) {
        dis[i] = new int[size];
        for (int j = 0; j <= G.n; j++) {
            adj[i*size + j] = dis[i][j] = INF;
        }
        adj[i*size + i] = dis[i][i] = 0;
        for (auto v : G.E[i]) {
            adj[i*size + v.first] = dis[i][v.first] = v.second;
        }
    }

    // Declare and Initialise Device Array
    int *devAdj; 
    allocCopy<int>(&devAdj, adj, size*size, "Adj");
    
    // Run Cuda Parallel
    dim3 blocks((size+THREADX-1)/THREADX, (size+THREADX-1)/THREADX, 1);
    dim3 threads(THREADX, THREADX, 1);
    hipEventRecord(start);
    for (int i = 0; i < size; i++) {
        APSP_kernel1<<< blocks, threads >>>(devAdj, i, size);
    }
    hipEventRecord(stop);
    cudaCheck(hipPeekAtLastError());
    if (cudaCheck(hipMemcpy(adj, devAdj, size*size*sizeof(int), hipMemcpyDeviceToHost))) {
        std::cout << "Obtained distance in host at adj" << std::endl;
    }

    // Calculate Time Taken
    hipEventSynchronize(stop);
    float timeGPU = 0;
    hipEventElapsedTime(&timeGPU, start, stop);
    std::cout << "CUDA Elapsed Time (in ms): " << timeGPU << std::endl;

    // ========================= CPU ============================= //
    auto beg = std::chrono::high_resolution_clock::now();
    floydWarshallCPU(G.n, dis);
    auto end = std::chrono::high_resolution_clock::now();
    float timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();
    std::cout << "CPU Elapsed Time (in ms): " << timeCPU / 1000 << std::endl;

    // ======================= Verification ==========================//
    for (int i = 0; i <= G.n; i++) {
        for (int j = 0; j <= G.n; j++) {
            if (dis[i][j] != adj[i*size + j]) {
                std::cout << "Not a Match at " << i << " " << j << std::endl;
                std::cout << "GPU dist: " << adj[i*size + j] << std::endl;
                std::cout << "CPU dist: " << dis[i][j] << std::endl;
                exit(EXIT_FAILURE);
            }   
        }
    }
}
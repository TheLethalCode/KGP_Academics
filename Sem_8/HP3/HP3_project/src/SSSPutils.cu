#include <hip/hip_runtime.h>


__global__ void SSSP_kernel1(int *V, int *E, int *W, bool *M, int *C, int *U, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n && M[tid]) {
        M[tid] = false;
        int pos = V[tid], size = E[pos];
        for (int i = pos + 1; i < pos + size + 1; i++) {
            int nid = E[i];
            atomicMin(&U[nid], C[tid] + W[i]);
        }
    }
}

__global__ void SSSP_kernel2(bool *M, int *C, int *U, bool *flag, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        if (C[tid] > U[tid]) {
            C[tid] = U[tid];
            M[tid] = true;
            *flag = true;
        }
        U[tid] = C[tid];
    }
}
#include "../include/core.h"
#include "../include/graph.h"
#include "../include/BFSutils.h"
#include "../include/algoCPU.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <utility>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 1024

int main(int argc, char* argv[]) {

    int s;
    Graph G(argc, argv);
    std::cout << "Source Vertex: ";
    std::cin >> s;
    // ========================= CUDA ============================= //
      
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Declare and Initialise Host Array
    int N, level, queueSize, nextQueueSize, Es;
    int *V, *E, *D, *P, *incrDegrees;
    
    N = vecToArr(G.posV, &V);
    Es = vecToArr(G.packE, &E);
    D = new int[N];
    P = new int[N];
    incrDegrees = new int[N];
    std::fill_n(D, N, INF);
    std::fill_n(P, N, INF);
    std::fill_n(incrDegrees, N, 0);
    D[s] = level = nextQueueSize = 0; // Update source values
    queueSize = 1;

    // Declare and Initialise Device Array
    int *devV, *devE, *devD, *devP;
    int *devCurrentQueue, *devNextQueue, *devDegrees, *devIncrDegrees;
    allocCopy<int>(&devV, V, N, "V_a");
    allocCopy<int>(&devE, E, Es, "E_a");
    allocCopy<int>(&devD, D, N, "D_a");
    allocCopy<int>(&devP, P, N, "P_a");
    alloc<int>(&devCurrentQueue, N, "devCurrentQueue");
    alloc<int>(&devNextQueue, N, "devNextQueue");
    alloc<int>(&devDegrees, N, "devDegrees");
    allocCopy<int>(&devIncrDegrees, incrDegrees, N, "devIncrDegrees");

    int firstElemQueue = s;
    hipMemcpy(devCurrentQueue, &firstElemQueue, sizeof(int), hipMemcpyHostToDevice);

    // Run Cuda Parallel
    hipEventRecord(start);
    while (queueSize) {
        int blocks = queueSize/NUM_THREADS + 1;
        nextLayer<<< blocks, NUM_THREADS >>>(level, devV, devE, devP, devD, queueSize, devCurrentQueue);
        hipDeviceSynchronize();
        countDegrees<<< blocks, NUM_THREADS >>>(devV, devE, devP, queueSize, devCurrentQueue, devDegrees);
        hipDeviceSynchronize();
        scanDegrees<<< blocks, NUM_THREADS >>>(queueSize, devDegrees, devIncrDegrees);
        hipDeviceSynchronize();
        hipMemcpy(incrDegrees, devIncrDegrees, sizeof(int)*N, hipMemcpyDeviceToHost);

        //count prefix sums on CPU for ends of blocks exclusive already written previous block sum
        incrDegrees[0] = 0;
        for (int i = NUM_THREADS; i < queueSize + NUM_THREADS; i += NUM_THREADS) {
            incrDegrees[i / NUM_THREADS] += incrDegrees[i / NUM_THREADS - 1];
        }
        nextQueueSize = incrDegrees[(queueSize - 1) / NUM_THREADS + 1];
        
        hipMemcpy(devIncrDegrees, incrDegrees, sizeof(int)*N, hipMemcpyHostToDevice);
        assignVerticesNextQueue<<< blocks, NUM_THREADS >>>(devV, devE, devP, queueSize, devCurrentQueue, devNextQueue,
            devDegrees, devIncrDegrees, nextQueueSize);
        hipDeviceSynchronize();
        level += 1;
        queueSize = nextQueueSize;
        std::swap(devCurrentQueue, devNextQueue);
    }
    hipEventRecord(stop);
    cudaCheck(hipPeekAtLastError());
    if (cudaCheck(hipMemcpy(D, devD, N * sizeof(int), hipMemcpyDeviceToHost))) {
        std::cout << "Obtained distance in host at D_a" << std::endl;
    }

    // Calculate Time Taken
    hipEventSynchronize(stop);
    float timeGPU = 0;
    hipEventElapsedTime(&timeGPU, start, stop);

    // Free memory
    clear<int>(devV, "devV");
    clear<int>(devE, "devE");
    clear<int>(devD, "devD");
    clear<int>(devP, "devP");
    clear<int>(devCurrentQueue, "devCurrentQueue");
    clear<int>(devNextQueue, "devNextQueue");
    clear<int>(devDegrees, "devDegrees");
    clear<int>(devIncrDegrees, "devIncrDegrees");

    // ========================= CPU ============================= //
    int *dis = new int[N];
    int *parent = new int[N];
    bool *visited = new bool[N];
    std::fill_n(dis, N, INF);
    std::fill_n(parent, N, -1);
    std::fill_n(visited, N, false);
    auto beg = std::chrono::high_resolution_clock::now();
    bfsCPU(s, G, dis, parent, visited);
    auto end = std::chrono::high_resolution_clock::now();
    float timeCPU = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();

    std::cout << "CUDA Elapsed Time (in ms): " << timeGPU << std::endl;
    std::cout << "CPU  Elapsed Time (in ms): " << timeCPU / 1000 << std::endl;

    // ======================= Verification ==========================//
    for (int i = 0; i < N; i++) {
        if (dis[i] != D[i]) {
            std::cout << "Not a Match at " << i << std::endl;
            std::cout << "GPU dist: " << D[i] << std::endl;
            std::cout << "CPU dist: " << dis[i] << std::endl;
            exit(EXIT_FAILURE);
        }
    }
}